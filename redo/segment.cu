#include "hip/hip_runtime.h"
#include "segment.cuh"

// helper function for determining if t in [a, b] where order of a, b is unknown

__device__ Segment::Segment() {}

Segment::Segment(Point* a, Point* b) : endpoint1(a), endpoint2(b) {}


double Segment::length() {

	double x1 = endpoint1->getX();
	double y1 = endpoint1->getY();
	double x2 = endpoint2->getX();
	double y2 = endpoint2->getY();
	return pow(pow(x1 - x2, 2) + pow(y1 - y2, 2), 0.5);
}

__device__ void Segment::unitNormal(double* nx, double* ny) {
	double deltaX = endpoint2->getX() - endpoint1->getX();
	double deltaY = endpoint2->getY() - endpoint1->getY();
	double unitX = deltaX / length();
	double unitY = deltaY / length();
	*nx = unitY;
	*ny = -unitX;
}

void Segment::scaledNormal(double* nx, double* ny) {
	double deltaX = endpoint2->getX() - endpoint1->getX();
	double deltaY = endpoint2->getY() - endpoint1->getY();
	*nx = deltaY / 2;
	*ny = -deltaX / 2;
}


__device__ void parametrize(Segment& e, Segment& f, double* t1, double* t2, double* det) {
	// parametrize and represent as matrix equation to be solved: 
	// t1 * x0 + (1-t1) * x1 = t2 * x2 + (1-t2) * x3
	// (x0-x1) * t1 + (x3-x2) * t2 = x3 - x1

	// note this originally was passed in to create a matrix
	// but dynamic matrix memory allocation costs time;
	// instead, use in this array form
	double arr[4];
	// first column (matches t1)
	arr[0] = e.endpoint1->getX() - e.endpoint2->getX();
	arr[2] = e.endpoint1->getY() - e.endpoint2->getY();
	// second column (matches t2)
	arr[1] = f.endpoint2->getX() - f.endpoint1->getX();
	arr[3] = f.endpoint2->getY() - f.endpoint1->getY();

	double determinant = arr[0] * arr[3] - arr[1] * arr[2];
	// target vector
	double targX = f.endpoint2->getX() - e.endpoint2->getX();
	double targY = f.endpoint2->getY() - e.endpoint2->getY();
	// scaled solution is adjugate of arr multiplied by target
	// adjugate is arr[3], -arr[1], -arr[2], arr[0]
	*t1 = arr[3] * targX - arr[1] * targY;
	*t2 = -arr[2] * targX + arr[0] * targY;
	*det = determinant;
}

__device__ bool isBetween(const double& t, const double& a, const double& b) {
	return (a <= t && t <= b) || (b <= t && t <= a);
}

__device__ bool Segment::intersection(Segment& other, Point* pt) {
	double t1 = 0;
	double t2 = 0;
	double det = 0;
	parametrize(*this, other, &t1, &t2, &det);
	bool detect = (det != 0) && isBetween(t1, 0, det) && isBetween(t2, 0, det);
	if (detect && pt) {
		double x = (endpoint1->getX() * t1 + endpoint2->getX() * (det - t1)) / det;
		double y = (endpoint1->getY() * t1 + endpoint2->getY() * (det - t1)) / det;
		*pt = Point(x, y);
	}
	return detect;
}


