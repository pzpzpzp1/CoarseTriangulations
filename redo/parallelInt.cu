#include "hip/hip_runtime.h"


#include "parallelInt.cuh"

// linear basis element derivatives
__device__ static const double phiU[3] = {-1, 1, 0};
__device__ static const double phiV[3] = {-1, 0, 1};

ParallelIntegrator::ParallelIntegrator() {
	threads2D = dim3(threadsX, threadsY);
}

bool ParallelIntegrator::initialize(Pixel *pix, int xMax, int yMax, ApproxType a, long long space, bool exact) {
	// steal references for easy access later
	pixArr = pix;
	approx = a;
	maxX = xMax;
	maxY = yMax;
	computeExact = exact;
	initialized = true;
	// allocate working computation space
	hipMallocManaged(&arr, approx * sizeof(double *));
	for(int i = 0; i < approx; i++) {
		hipMallocManaged(&(arr[i]), space * sizeof(double));
	}
	// less space needed for helper because it is only used for summing arr
	long long helperSpace = ceil(space / 512.0);
	hipMallocManaged(&helper, helperSpace * sizeof(double));
	// the above operations may cause errors because so much memory is required
	hipError_t error = hipGetLastError();
  	if(error != hipSuccess) {
		printf("CUDA error: %s\n", hipGetErrorString(error));
		cout << "An approximation of this quality is not possible due to memory limitations." << endl;
		return false;
	}
	hipMallocManaged(&curTri, 3 * sizeof(Point));
	if(hipGetLastError() != hipSuccess) {
		printf("CUDA error: %s\n", hipGetErrorString(error));
		return false;
	}
	return true;
}

ParallelIntegrator::~ParallelIntegrator() {
	if(initialized) {
		for(int i = 0; i < approx; i++) {
			hipFree(arr[i]);
		}
		hipFree(arr);
		hipFree(helper);
		hipFree(curTri);
	}
}

__device__ void warpReduce(volatile double *sdata, unsigned int tid) {
	sdata[tid] += sdata[tid + 32];
	sdata[tid] += sdata[tid + 16];
	sdata[tid] += sdata[tid + 8];
	sdata[tid] += sdata[tid + 4];
	sdata[tid] += sdata[tid + 2];
	sdata[tid] += sdata[tid + 1];
}

// kernel for sumArray
// compute the sum of an array arr with given size, in parallel
// with 1D thread/blocks, storing the result per block in result
__global__ void sumBlock(double *arr, int size, double *result) {
	__shared__ double partial[1024]; // hold partial results
	int tid = threadIdx.x;
	int ind = blockIdx.x * 2 * blockDim.x + tid;
	// load into partial result array
	if(ind + blockDim.x < size) {
		partial[tid] = arr[ind] + arr[ind + blockDim.x];
	} else if(ind < size) {
		partial[tid] = arr[ind];
	} else {
		partial[tid] = 0;
	}
	__syncthreads();

	// completely unroll the reduction
	if(tid < 512) {
		partial[tid] += partial[tid + 512];
	}
	__syncthreads();
	if(tid < 256) {
		partial[tid] += partial[tid + 256];
	}
	__syncthreads();
	if(tid < 128) {
		partial[tid] += partial[tid + 128];
	}
	__syncthreads();
	if(tid < 64) {
		partial[tid] += partial[tid + 64];
	}
	__syncthreads();

	// only one active warp at this point
	if(tid < 32) {
		warpReduce(partial, tid);
	}

	// write output for block to result
	if(tid == 0) {
		result[blockIdx.x] = partial[0];
	}
}

double ParallelIntegrator::sumArray(int size, int i) {
	int curSize = size; // current length of array to sum
	int numBlocks = (size + 2 * threads1D - 1) / (2 * threads1D);
	bool ansArr = true; // whether results are currently held in arr
	while(curSize > 1) {
		if(ansArr) {
			sumBlock<<<numBlocks, threads1D>>>(arr[i], curSize, helper);
		} else {
			sumBlock<<<numBlocks, threads1D>>>(helper, curSize, arr[i]);
		}
		hipDeviceSynchronize();
		curSize = numBlocks;
		numBlocks = (numBlocks + 2 * threads1D - 1) / (2 * threads1D);
		ansArr = !ansArr;
	}
	// at this point the array has been summed
	if(ansArr) { // arr should hold the results
		return arr[i][0];
	}
	return helper[0];
}

// kernel for constantEnergyEval
// compute the energy of a single pixel on triangle triArr[t]
// weight by saliency value of pixel if salient
template<bool salient>
__global__ void pixConstantEnergyInt(Pixel *pixArr, int maxX, int maxY, Triangle tri, double color, double *results) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int ind = x * maxY + y; // index in pixArr;
	if(x < maxX && y < maxY) {
		double area = pixArr[ind].intersectionArea(tri);
		if(salient) area *= pixArr[ind].getSaliency();
		double diff = color - pixArr[ind].getColor();
		results[ind] = diff * diff * area;
	}
}

double ParallelIntegrator::constantEnergyExact(Triangle *tri, double color, bool salient) {
	dim3 numBlocks((maxX + threadsX - 1) / threadsX, (maxY + threadsY - 1) / threadsY);
	if(salient) {
		pixConstantEnergyInt<true><<<numBlocks, threads2D>>>(pixArr, maxX, maxY, *tri, color, arr[0]);
	} else {
		pixConstantEnergyInt<false><<<numBlocks, threads2D>>>(pixArr, maxX, maxY, *tri, color, arr[0]);
	}
	double answer = sumArray(maxX * maxY);
	return answer;
}

// kernel for constant energy approx
// using Point a as vertex point, sample ~samples^2/2 points inside the triangle with a triangular area element of dA
// NOTE: samples does not count endpoints along edge bc as the parallelograms rooted there lie outside the triangle
// maxY is for converting 2D pixel index to 1D index
template<bool salient>
__global__ void approxConstantEnergySample(Pixel *pixArr, int maxX, int maxY, Point *a, Point *b, Point *c, double color, double *results, double dA, int samples) {
	int u = blockIdx.x * blockDim.x + threadIdx.x; // component towards b
	int v = blockIdx.y * blockDim.y + threadIdx.y; // component towards c
	int ind = (2 * samples - u + 1) * u / 2 + v; // 1D index in results
	// this is because there are s points in the first column, s-1 in the next, etc. up to s - u + 1
	if(u + v < samples) {
		// get coordinates of this point using appropriate weights
		double x = (a->getX() * (samples - u - v) + b->getX() * u + c->getX() * v) / samples;
		double y = (a->getY() * (samples - u - v) + b->getY() * u + c->getY() * v) / samples;
		// find containing pixel
		int pixX = pixelRound(x, maxX);
		int pixY = pixelRound(y, maxY);
		double diff = color - pixArr[pixX * maxY + pixY].getColor();
		// account for points near edge bc having triangle contributions rather than parallelograms,
		// written for fast access and minimal branching
		double areaContrib = (u + v == samples - 1) ? dA : 2 * dA;
		if(salient) areaContrib *= pixArr[pixX * maxY + pixY].getSaliency();
		results[ind] = diff * diff * areaContrib;
	}
}

double ParallelIntegrator::constantEnergyApprox(Triangle *tri, double color, double ds, bool salient) {
	int i = tri->midVertex(); // vertex opposite middle side
	// ensure minVertex is copied into location curTri
	tri->copyVertices(curTri+((3-i)%3), curTri+((4-i)%3), curTri+((5-i)%3));
	// compute number of samples needed, using median number per side
	int samples = ceil(curTri[1].distance(curTri[2])/ds);
	// unfortunately half of these threads will not be doing useful work; no good fix, sqrt is too slow for triangular indexing
	dim3 numBlocks((samples + threadsX - 1) / threadsX, (samples + threadsY - 1) / threadsY);
	double dA = tri->getArea() / (samples * samples);
	if(salient) {
		approxConstantEnergySample<true><<<numBlocks, threads2D>>>(pixArr, maxX, maxY, curTri, curTri+1, curTri+2, color, arr[0], dA, samples);
	} else {
		approxConstantEnergySample<false><<<numBlocks, threads2D>>>(pixArr, maxX, maxY, curTri, curTri+1, curTri+2, color, arr[0], dA, samples);
	}
	hipDeviceSynchronize();
	double answer = sumArray(samples * (samples + 1) / 2);
	return answer;
}

double ParallelIntegrator::constantEnergyEval(Triangle *tri, double color, double ds, bool salient) {
	// switch integration method based on exactnes required
	if(computeExact) {
		return constantEnergyExact(tri, color, salient);
	}
	return constantEnergyApprox(tri, color, ds, salient);
}

// kernel for constant line integral exact evaluation
// compute line integral of v dot n f ds for a single pixel and single triangle a, b, c when point b is moving
__global__ void pixConstantLineInt(Pixel *pixArr, int maxX, int maxY, Point *a, Point *b, Point *c, bool isX, double *results) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int ind = x * maxY + y;
	if (x < maxX && y < maxY) {
		double answer = 0;
		for(int i = 0; i < 2; i++) { // v dot n is nonzero only on a -- b and b -- c
			// extract segment and maintain ccw order for outward normal
			Segment seg = (i == 0) ? Segment(a, b) : Segment(b, c);
			Point *segEnd = (i == 0) ? a : c; // determine endpoint of seg that is not b
			double midX, midY; // to hold midpoint of segment intersection with this pixel
			double length = pixArr[ind].intersectionLength(seg, &midX, &midY);
			if(length != 0) {
				Point midpoint(midX, midY);
				// compute velocity at this point by scaling
				double distanceToVertex = midpoint.distance(*segEnd);
				double scale = distanceToVertex / seg.length(); // 1 if at b, 0 at opposite edge
				// velocity components
				double velX = (isX) ? scale : 0;
				double velY = scale - velX;
				// get unit normal values for this segment
				double nx, ny;
				seg.unitNormal(&nx, &ny);
				double vn = velX * nx + velY * ny; // average value of v * n
				answer += vn * length * pixArr[ind].getColor();
			}
		}
		results[ind] = answer;
	}
}

double ParallelIntegrator::lineIntExact(Triangle *tri, int pt, bool isX) {
	dim3 numBlocks((maxX + threadsX - 1) / threadsX, (maxY + threadsY - 1) / threadsY);
	tri->copyVertices(curTri, curTri+1, curTri+2);
	// compute integral in parallel based on function to integrate
	switch (approx) {
		case constant: {
			pixConstantLineInt<<<numBlocks, threads2D>>>(pixArr, maxX, maxY, curTri+((pt+2)%3), curTri+pt, curTri+((pt+1)%3), isX, arr[0]);
			break;
		}
		case linear: 
			cout << "Exact integration on linear approximations is not supported." << endl;
			exit(EXIT_FAILURE);
			break;
		case quadratic: // TODO
			break;
	}
	double answer = sumArray(maxX * maxY);
	return answer;
}

// kernel for constant line integral approximation
// compute line integral of v dot n f ds where point a is moving; 
// reverse determines if integral should be computed from a to b (false) or opposite
__global__ void constLineIntSample(Pixel *pixArr, int maxX, int maxY, Point *a, Point *b, bool reverse, bool isX, double *results, double ds, int samples) {
	int k = blockIdx.x * blockDim.x + threadIdx.x; // index along a to b
	if(k < samples) {
		// extract current point and containing pixel
		double x = (a->getX() * (samples - k) + b->getX() * k) / samples;
		double y = (a->getY() * (samples - k) + b->getY() * k) / samples;
		int pixX = pixelRound(x, maxX);
		int pixY = pixelRound(y, maxY);
		// velocity components
		double scale = ((double) samples - k) / samples; // 1 when k = 0 (evaluate at a) and 0 at b
		double velX = (isX) ? scale : 0;
		double velY = scale - velX;
		// extract unit normal, manually for the sake of speed
		double length = a->distance(*b); // length of whole segment
		// assume going from a to b first, want normal pointing right
		double nx = (b->getY() - a->getY()) / length;
		double ny = (a->getX() - b->getX()) / length;
		double vn = velX * nx + velY * ny; // value of v * n at this point
		// flip vn if normal is actually pointing the other way (integrate from b to a)
		if(reverse) vn *= -1;
		results[k] = vn * ds * pixArr[pixX * maxY + pixY].getColor();
	}
}

// kernel for constant line integral approximation, by taking average of all sample points
// phiA indicates whether the basis element at a is being integrated;
// if false, integrate element at b (element at c is zero on this segment)
__global__ void linearLineIntSample(Pixel *pixArr, int maxX, int maxY, Point *a, Point *b, bool reverse, bool isX, double *results, int samples, bool phiA) {
	int k = blockIdx.x * blockDim.x + threadIdx.x; // index along a to b
	if(k <= samples) {
		// extract current point and containing pixel
		double x = (a->getX() * (samples - k) + b->getX() * k) / samples;
		double y = (a->getY() * (samples - k) + b->getY() * k) / samples;
		int pixX = pixelRound(x, maxX);
		int pixY = pixelRound(y, maxY);
		// velocity components
		double scale = ((double) samples - k) / samples; // 1 when k = 0 (evaluate at a) and 0 at b
		double velX = (isX) ? scale : 0;
		double velY = scale - velX;
		// extract unit normal, manually for the sake of speed
		double length = a->distance(*b); // length of whole segment
		// assume going from a to b first, want normal pointing right
		double nx = (b->getY() - a->getY()) / length;
		double ny = (a->getX() - b->getX()) / length;
		double vn = velX * nx + velY * ny; // value of v * n at this point
		// flip vn if normal is actually pointing the other way (integrate from b to a)
		if(reverse) vn *= -1;
		// get value of phi
		double phi = (phiA) ? scale : 1 - scale; // since phi_j is linear, corresponds to scale
		results[k] = pixArr[pixX * maxY + pixY].getColor() * phi * vn;
	}
}

double ParallelIntegrator::lineIntApprox(Triangle *tri, int pt, bool isX, double ds, int basisInd) {
	// ensure pt is copied into the first slot of curTri
	tri->copyVertices(curTri+((3-pt)%3), curTri+((4-pt)%3), curTri+((5-pt)%3));
	// get number of samples for side pt, pt+1 and side pt, pt+2
	int samples[2];
	int numBlocks[2];
	for(int i = 0; i < 2; i++) {
		samples[i] = ceil(curTri->distance(curTri[i+1])/ds);
		numBlocks[i] = ceil(1.0 * samples[i] / threads1D);
	};
	double answer = 0; // integrate over both moving sides
	switch(approx) {
		case constant: {
			for(int i = 0; i < 2; i++) {
				double totalLength = curTri->distance(curTri[i+1]);
				// actual dx being used
				double dx = totalLength / samples[i];
				constLineIntSample<<<numBlocks[i], threads1D>>>(pixArr, maxX, maxY, curTri, curTri+i+1, (i==1), isX, arr[0], dx, samples[i]);
				hipError_t cudaStatus = hipDeviceSynchronize();
				answer += sumArray(samples[i]);
				cudaStatus = hipDeviceSynchronize();
			}
			break;
		}
		case linear:
			// v phi_j is nonzero only if the line contains both vertex pt and basisInd
			if(basisInd == pt) {
				for(int i = 0; i < 2; i++) {
					double totalLength = curTri->distance(curTri[i+1]);
					// in case num samples is too small; ensure at least 2 points are sampled
					// (also prevent zero division error)
					samples[i] = max(samples[i], 2);
					linearLineIntSample<<<numBlocks[i], threads1D>>>(pixArr, maxX, maxY, curTri, curTri+i+1, (i==1), isX, arr[0], samples[i]-1, true);
					answer += totalLength * sumArray(samples[i]) / samples[i];
				}
			} else { // integrate along segment pt, basisInd
				int offset = (basisInd - pt + 3) % 3; // index of basisInd relative to pt
				int i = (offset + 1)%2; // index for this side's data in samples and numBlocks
				double totalLength = curTri->distance(curTri[offset]);
				// ensure at least 2 points are sampled
				samples[i] = max(samples[i], 2);
				linearLineIntSample<<<numBlocks[i], threads1D>>>(pixArr, maxX, maxY, curTri, curTri+offset, (offset==2), isX, arr[0], samples[i]-1, false);
				answer += totalLength * sumArray(samples[i]) / samples[i];
			}
			break;
		case quadratic:
			break;
	}
	return answer;
}

double ParallelIntegrator::lineIntEval(Triangle *tri, int pt, bool isX, double ds, int basisInd) {
	if(computeExact) {
		return lineIntExact(tri, pt, isX);
	}
	return lineIntApprox(tri, pt, isX, ds, basisInd);
}

// kernel for exact double integral
// compute double integral of f dA for a single pixel and single triangle triArr[t]
// pixArr is a 1D representation of image, where pixel (x, y) is at x * maxY + y
// reults holds the result for each pixel
__global__ void pixConstantDoubleInt(Pixel *pixArr, int maxX, int maxY, Triangle tri, double *results, ColorChannel channel) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int ind = x * maxY + y; // index in pixArr
	if(x < maxX && y < maxY) { // check bounds
		double area = pixArr[ind].intersectionArea(tri);
		results[ind] = area * pixArr[ind].getColor(channel);
	}
}

double ParallelIntegrator::doubleIntExact(Triangle *tri, ColorChannel channel) {
	dim3 numBlocks((maxX + threadsX -1) / threadsX, (maxY + threadsY -1) / threadsY);
	// compute integral in parallel based on function to integrate
	switch (approx) {
		case constant: {
			pixConstantDoubleInt<<<numBlocks, threads2D>>>(pixArr, maxX, maxY, *tri, arr[0], channel);
			break;
		}
		case linear: // TODO: fill out
			cout << "Exact integrals on non-constant approximations are not supported. Please change your approximation type." << endl;
			exit(EXIT_FAILURE);
			break;
		case quadratic: // TODO: fill out
			break;
	}
	double answer = sumArray(maxX * maxY);
	return answer;
}

// kernel for double integral approximation
// using Point a as vertex point, sample ~samples^2/2 points inside triangle with area element of dA
// for details see approxConstantEnergySample above
__global__ void constDoubleIntSample(Pixel *pixArr, int maxX, int maxY, Point *a, Point *b, Point *c, double *results, double dA, int samples, ColorChannel channel) {
	int u = blockIdx.x * blockDim.x + threadIdx.x; // component towards b
	int v = blockIdx.y * blockDim.y + threadIdx.y; // component towards c
	int ind = (2 * samples - u + 1) * u / 2 + v; // 1D index in results
	if(u + v < samples) {
		double x = (a->getX() * (samples - u - v) + b->getX() * u + c->getX() * v) / samples;
		double y = (a->getY() * (samples - u - v) + b->getY() * u + c->getY() * v) / samples;
		// find containing pixel
		int pixX = pixelRound(x, maxX);
		int pixY = pixelRound(y, maxY);
		double areaContrib = (u+v == samples - 1) ? dA : 2 * dA;
		results[ind] = pixArr[pixX * maxY + pixY].getColor(channel) * areaContrib;
	}
}

// compute integral f phi_j dA by barycentric sampling
// using pts[0] as vertex point; store values in results[j]
__global__ void linearDoubleIntSample(Pixel *pixArr, int maxX, int maxY, Point *pts, double **results, double dA, int samples, ColorChannel channel) {
	int u = blockIdx.x * blockDim.x + threadIdx.x; // component towards pts[1]
	int v = blockIdx.y * blockDim.y + threadIdx.y; // component towards pts[2]
	int ind = (2 * samples - u + 1) * u / 2 + v; // 1D index in results[j]
	if(u + v < samples) {
		// extract coordinates at this sample point
		double x = (pts[0].getX() * (samples - u - v) + pts[1].getX() * u + pts[2].getX() * v) / samples;
		double y = (pts[0].getY() * (samples - u - v) + pts[1].getY() * u + pts[2].getY() * v) / samples;
		// get color of containing pixel
		double color = pixArr[pixelRound(x, maxX) * maxY + pixelRound(y, maxY)].getColor(channel);
		// scale fdA by 1/samples to avoid multiple divisions in FEM basis computation later
		double fdA = dA * color / samples;
		// area element is a parallelogram except for triangular contributions at the opposite edge
		// when u + v == samples - 1
		if(u + v < samples - 1) fdA *= 2;
		// second factor is (scaled) FEM basis value at this point
		results[0][ind] = fdA * (samples - u - v);
		results[1][ind] = fdA * u;
		results[2][ind] = fdA * v;
	}
}

void ParallelIntegrator::doubleIntApprox(Triangle *tri, double ds, double *result, ColorChannel channel) {
	// extract number of samples
	int i = tri->midVertex();
	// copy middle vertex into curTri[0]
	tri->copyVertices(curTri+((3-i)%3), curTri+((4-i)%3), curTri+((5-i)%3));
	int samples = ceil(curTri[1].distance(curTri[2])/ds);
	dim3 numBlocks((samples + threadsX - 1) / threadsX, (samples + threadsY - 1) / threadsY);
	double dA = tri->getArea() / (samples * samples);
	switch(approx) {
		case constant: {
			constDoubleIntSample<<<numBlocks, threads2D>>>(pixArr, maxX, maxY, curTri, curTri+1, curTri+2, arr[0], dA, samples, channel);
			break;
		}
		case linear: {
			linearDoubleIntSample<<<numBlocks, threads2D>>>(pixArr, maxX, maxY, curTri, arr, dA, samples, channel);
			break;
		}
	}
	hipDeviceSynchronize();

	// store results into result in order aligning with tri
	// (result[j] is integral of phi_j, which is 1 on tri.vertices[j] and 0 on the other vertices)
	for(int j = 0; j < approx; j++) {
		int relativeBasis = (j + approx - i) % approx; // with reference to vertices of curTri; when j == i, this is 0
		result[j] = sumArray(samples * (samples + 1) / 2, relativeBasis);
	}
}

void ParallelIntegrator::doubleIntEval(Triangle *tri, double ds, double *result, ColorChannel channel) {
	if(computeExact) {
		*result = doubleIntExact(tri, channel);
	} else {
		doubleIntApprox(tri, ds, result, channel);
	}
}

// kernel function for linearEnergyApprox
// assuming point a as vertex and matching k0, k1, k2 to a, b, c,
// sample (f - sum k_i phi_i)^2 over the triangle
// weighted by saliency if salient
template<bool salient>
__global__ void approxLinearEnergySample(Pixel *pixArr, int maxX, int maxY, Point *a, Point *b, Point *c, double k0, double k1, double k2, double *results, double dA, int samples) {
	int u = blockIdx.x * blockDim.x + threadIdx.x; // component towards b
	int v = blockIdx.y * blockDim.y + threadIdx.y; // component towards c
	int ind = (2 * samples - u + 1) * u / 2 + v; // 1D index in results
	// this is because there are s points in the first column, s-1 in the next, etc. up to s - u + 1
	if(u + v < samples) {
		// get coordinates of this point using appropriate weights
		double x = (a->getX() * (samples - u - v) + b->getX() * u + c->getX() * v) / samples;
		double y = (a->getY() * (samples - u - v) + b->getY() * u + c->getY() * v) / samples;
		// find containing pixel
		int pixX = pixelRound(x, maxX);
		int pixY = pixelRound(y, maxY);
		// find color at this point using standard transform
		double diff = (k0 * (samples - u - v) + k1 * u + k2 * v) / samples - pixArr[pixX * maxY + pixY].getColor();
		// account for points near edge bc having triangle contributions rather than parallelograms,
		// written for fast access and minimal branching
		double areaContrib = (u + v == samples - 1) ? dA : 2 * dA;
		if(salient) {
			areaContrib *= pixArr[pixX * maxY + pixY].getSaliency();
		}
		results[ind] = diff * diff * areaContrib;
	}
}

double ParallelIntegrator::linearEnergyApprox(Triangle *tri, double *coeffs, double ds, bool salient) {
	int i = tri->midVertex(); // vertex opposite middle side
	// curTri[0] = tri.vertices[i]
	tri->copyVertices(curTri + ((3-i)%3), curTri + ((4-i)%3), curTri + ((5-i)%3));
	// compute number of samples needed, using median number per side
	int samples = ceil(curTri[1].distance(curTri[2])/ds);
	// unfortunately half of these threads will not be doing useful work; no good fix, sqrt is too slow for triangular indexing
	dim3 numBlocks((samples + threadsX - 1) / threadsX, (samples + threadsY - 1) / threadsY);
	double dA = tri->getArea() / (samples * samples);
	if(salient) {
		approxLinearEnergySample<true><<<numBlocks, threads2D>>>(pixArr, maxX, maxY, curTri, curTri + 1, curTri + 2,
			coeffs[i], coeffs[(i+1)%3], coeffs[(i+2)%3], arr[0], dA, samples);
	} else {
		approxLinearEnergySample<false><<<numBlocks, threads2D>>>(pixArr, maxX, maxY, curTri, curTri + 1, curTri + 2,
			coeffs[i], coeffs[(i+1)%3], coeffs[(i+2)%3], arr[0], dA, samples);
	}
	double answer = sumArray(samples * (samples + 1) / 2);
	return answer;
}

// kernel function for computing integral of 2A_T f d(phi_j) dA when pts[0] is moving at (1,0)
// phiU, phiV indicate d phi/du, d phi/dv and dA_x is the area gradient
__global__ void linearImageGradientX(Pixel *pixArr, int maxX, int maxY, Point *pts, double **results, double dA, double dA_x, int samples) {
	int uInd = blockIdx.x * blockDim.x + threadIdx.x; // component towards pts[1]
	int vInd = blockIdx.y * blockDim.y + threadIdx.y; // component towards pts[2]
	double u = (double) uInd / samples;
	double v = (double) vInd / samples;
	int ind = (2 * samples - uInd + 1) * uInd / 2 + vInd; // 1D index in results
	if(uInd + vInd < samples) {
		// get coordinates of this point using appropriate weights
		double x = (pts[0].getX() * (samples - uInd - vInd) + pts[1].getX() * uInd + pts[2].getX() * vInd) / samples;
		double y = (pts[0].getY() * (samples - uInd - vInd) + pts[1].getY() * uInd + pts[2].getY() * vInd) / samples;
		// compute du/dt, dv/dt at this point (scaled by 2A_T)
		double du = y - pts[2].getY() - 2 * u * dA_x;
		double dv = pts[1].getY() - y - 2 * v * dA_x;
		// find color of containing pixel
		double color = pixArr[pixelRound(x, maxX) * maxY + pixelRound(y, maxY)].getColor();
		// account for points near opposite edge having triangle contributions rather than parallelograms
		double fdA = (uInd + vInd == samples - 1) ? color * dA : 2 * color * dA;
		// compute all three basis element contributions
		results[0][ind] = fdA * (phiU[0] * du + phiV[0] * dv);
		results[1][ind] = fdA * (phiU[1] * du + phiV[1] * dv);
		results[2][ind] = fdA * (phiU[2] * du + phiV[2] * dv);
	}
}

// same kernel function but when pts[0] is moving at (0,1)
__global__ void linearImageGradientY(Pixel *pixArr, int maxX, int maxY, Point *pts, double **results, double dA, double dA_y, int samples) {
	int uInd = blockIdx.x * blockDim.x + threadIdx.x; // component towards pts[1]
	int vInd = blockIdx.y * blockDim.y + threadIdx.y; // component towards pts[2]
	double u = (double) uInd / samples;
	double v = (double) vInd / samples;
	int ind = (2 * samples - uInd + 1) * uInd / 2 + vInd; // 1D index in results
	if(uInd + vInd < samples) {
		// get coordinates of this point using appropriate weights
		double x = (pts[0].getX() * (samples - uInd - vInd) + pts[1].getX() * uInd + pts[2].getX() * vInd) / samples;
		double y = (pts[0].getY() * (samples - uInd - vInd) + pts[1].getY() * uInd + pts[2].getY() * vInd) / samples;
		// find du/dt, dv/dt at this point (scaled by 2A_T)
		double du = pts[2].getX() - x - 2 * u * dA_y;
		double dv = x - pts[1].getX() - 2 * v * dA_y;
		// find color of containing pixel
		double color = pixArr[pixelRound(x, maxX) * maxY + pixelRound(y, maxY)].getColor();
		// account for points near opposite edge having triangle contributions rather than parallelograms
		double fdA = (uInd + vInd == samples - 1) ? color * dA : 2 * color * dA;
		// compute all three basis element contributions
		results[0][ind] = fdA * (phiU[0] * du + phiV[0] * dv);
		results[1][ind] = fdA * (phiU[1] * du + phiV[1] * dv);
		results[2][ind] = fdA * (phiU[2] * du + phiV[2] * dv);
	}
}

void ParallelIntegrator::linearImageGradient(Triangle *tri, int pt, bool isX, double ds, double *result) {
	// copy pt into curTri[0]
	tri->copyVertices(curTri+((3-pt)%3), curTri+((4-pt)%3), curTri+((5-pt)%3));
	// extract number of samples
	int i = tri->midVertex();
	int samples = ceil(((tri->vertices[(i+1)%3])->distance(*(tri->vertices[(i+2)%3])))/ds);
	dim3 numBlocks((samples + threadsX - 1) / threadsX, (samples + threadsY - 1) / threadsY);
	double dA = tri->getArea() / (samples * samples);
	double dA_x = tri->gradX(pt);
	double dA_y = tri->gradY(pt);
	if(isX) {
		linearImageGradientX<<<numBlocks, threads2D>>>(pixArr, maxX, maxY, curTri, arr, dA, dA_x, samples);
	} else {
		linearImageGradientY<<<numBlocks, threads2D>>>(pixArr, maxX, maxY, curTri, arr, dA, dA_y, samples);
	}
	for(int j = 0; j < approx; j++) {
		int relativeBasis = (j - pt + approx) % approx; // align curTri with ordering of basis elements
		result[j] = sumArray(samples * (samples + 1) / 2, relativeBasis) / (2 * tri->getArea());
	}
}