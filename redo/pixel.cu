#include "hip/hip_runtime.h"
#include "pixel.cuh"

// constants for converting rgb to grayscale
const double RED_LUMINANCE = 0.2126;
const double GREEN_LUMINANCE = 0.7152;
const double BLUE_LUMINANCE = 0.0722;

// get luminance of an rgb value by standard transformation
int getLuminance(int r, int g, int b) {
	return round(r * RED_LUMINANCE + g * GREEN_LUMINANCE + b * BLUE_LUMINANCE);
}

// helper functions

// determine whether a value has fractional part 1/2
// (used to determine whether point is a pixel corner)
__device__ bool isHalfInteger(double x) {
	return (x - floor(x) == 0.5);
}

// determine whether two points are "essentially" equal (floating point error)
__device__ bool approxEqual(Point &a, Point &b, double tolerance = 1e-12) {
	return (a.distance(b) < tolerance);
}

// compute (unsigned) area of the polygon enclosed by points,
// where edegs of the polygon are given by points[i] -- points[i+1]
__device__ double shoelace(Point *points, int &size) {
	if (size < 3) {
		return 0;
	}
	double area = 0;
	for(int i = 0; i < size; i++) {
		double x0 = points[i].getX();
		double y0 = points[i].getY();
		double x1 = points[(i+1)%size].getX();
		double y1 = points[(i+1)%size].getY();
		area += (x0 * y1 - x1 * y0);
	}
	// in practice points is ccw
	// up to floating point errors that don't affect area
	//assert(area >= 0);
	return area/2;
}

// compute integral of x over polygon points and store it in totalX, sim for y
// center is a reference point inside the pixel; even if it lies outside the polygon,
// using signed areas means the result will still be correct
__device__ void integrateXY(double *totalX, double *totalY, Point *points, int &size, Point &center) {
	double sumX = 0;
	double sumY = 0;
	for(int i = 0; i < size; i++) {
		// average value over a triangle is just the centroid
		double centroidX = (points[i].getX() + points[(i+1)%size].getX() + center.getX())/3;
		double centroidY = (points[i].getY() + points[(i+1)%size].getY() + center.getY())/3;
		double triangleArea = Triangle::getSignedArea(&center, &points[i], &points[(i+1)%size]);
		// weight the average
		sumX += centroidX * triangleArea;
		sumY += centroidY * triangleArea;
	}
	*totalX = sumX;
	*totalY = sumY;
}

// compute average values of x, y over the polygon enclosed by points
// and put them in the given variables
// center is again a reference point
__device__ void averageXY(double *avgX, double *avgY, Point *points, int &size, Point &center) {
	double totalX;
	double totalY;
	integrateXY(&totalX, &totalY, points, size, center);
	double totalArea = shoelace(points, size);
	*avgX = totalX / totalArea;
	*avgY = totalY / totalArea;
}

Pixel::Pixel(int x_, int y_, int c) : x(x_), y(y_) {
	corners[0] = Point(x-0.5, y-0.5);
	corners[1] = Point(x+0.5, y-0.5);
	corners[2] = Point(x+0.5, y+0.5);
	corners[3] = Point(x-0.5, y+0.5);
	for(int i = 0; i < 4; i++) {
		colors[i] = c;
	}
}

Pixel::Pixel(int x_, int y_, int r, int g, int b) : x(x_), y(y_) {
	corners[0] = Point(x-0.5, y-0.5);
	corners[1] = Point(x+0.5, y-0.5);
	corners[2] = Point(x+0.5, y+0.5);
	corners[3] = Point(x-0.5, y+0.5);
	colors[0] = r;
	colors[1] = g;
	colors[2] = b;
	colors[3] = getLuminance(r, g, b);
}

double Pixel::getColor(ColorChannel channel) {
	return colors[channel];
}

__device__ double Pixel::getSaliency() {
	return saliency;
}

void Pixel::setSaliency(double s) {
	assert(s >= 0);
	saliency = s;
}

__device__ bool Pixel::containsPoint(Point &p) {
	double px = p.getX();
	double py = p.getY();
	return (-0.5+x <= px && px <= 0.5+x) && (-0.5+y <= py && py <= 0.5+y);
}

__device__ double Pixel::intersectionLength(Segment &e, double *xVal, double *yVal) {
	Point intersections[2]; // hold intersections
	int numPts = 0; // track number of intersection points detected thus far
	Point intersectionPoint; // hold current potential intersection point
	for(int i = 0; i < 4; i++) {
		// retrieve a side of the pixel; at most two will have an 
		// intersection unless intersection is at corners
		Segment side(&corners[i], &corners[(i+1)%4]);
		bool collision = side.intersection(e, &intersectionPoint);
		if (collision) {
			bool isNewPoint = true; // whether this intersection is a new distinct point
			for(int i = 0; i < numPts; i++) {
				if(approxEqual(intersections[i], intersectionPoint)) {
					isNewPoint = false;
				}
			}
			if (isNewPoint) {
				intersections[numPts] = intersectionPoint;
				numPts++;
			}
		}
	}
	// handle segment endpoints potentially inside the pixel
	if (numPts < 2) {
		Point start = *(e.endpoint1);
		Point end = *(e.endpoint2);
		if (containsPoint(start)) {
			intersections[numPts] = start;
			numPts++;
		}
		if (containsPoint(end)) {
			intersections[numPts] = end;
			numPts++;
		}
	}
	if (numPts < 2) {
		return 0;
	}
	Segment contained(&intersections[0], &intersections[1]);
	// check for null pointers, assign midpoint coords
	if (xVal && yVal) {
		*xVal = (intersections[0].getX() + intersections[1].getX())/2;
		*yVal = (intersections[0].getY() + intersections[1].getY())/2;
	}
	return contained.length();
}

__device__ double Pixel::intersectionArea(Triangle t, Point* polygon, int *size) {
	Point center(x, y); // center of this pixel
	int numPoints = 0; // track number of points in polygon
	Point boundary[8]; // there should only be max 8 points on the boundary,
	int inInd; // index of some triangle vertex that lies inside pixel (may not exist)
	Segment triangleSides[3]; // hold sides of triangle

	// goal: compute boundary of the intersection

	for(int i = 0; i < 3; i++) {
		triangleSides[i] = Segment(t.vertices[i], t.vertices[(i+1)%3]);
		// add triangle vertices which may be inside the pixel, but don't add corners
		bool isCorner = isHalfInteger(t.vertices[i]->getX()) && isHalfInteger(t.vertices[i]->getY());
        if (!isCorner && containsPoint(*(t.vertices[i]))) {
            inInd = i;
			boundary[numPoints] = *(t.vertices[i]);
			numPoints++;
		}
	}

    // determine corner to start so as to preserve ccw property
    int start = 0;
    // do this by starting from a corner outside the triangle (if it exists);
	// if it doesn't exist start will stay at 0
    for(int i = 0; i < 4; i++) {
        // additionally, if there is exactly one point inside the triangle, make sure to start
        // at a corner on the same side of the interior point so that the first edge
        // interior point -- intersection point is correct (avoid issues of pixel corners inside
        // the triangle being non-adjacent)
        bool safelyOriented = (numPoints != 1) || 
			(Triangle::getSignedArea(corners + i, t.vertices[(inInd+1)%3], t.vertices[(inInd+2)%3]) >= 0);
        if (safelyOriented && !t.contains(corners[i])) {
			start = i;
			break; // including this line gives a 25% speed increase
		}
	}
    for(int i = 0; i < 4; i++) {
        // first determine if corner of pixel is inside
        Point corner = corners[(i+start) % 4];
		Segment side(corners + ((i+start)%4), corners + ((i+start+1)%4));
		// OPTIMIZATION: BRANCHING HERE; unavoidable?
        if (t.contains(corner)) {
			boundary[numPoints] = corner;
			numPoints++;
		}
        // determine intersections with side (i, i+1)
		Point sideIntersections[2];
		int intersectNum = 0; // track index in sideIntersections
		Point intersectionPoint; // track current intersection point
        for(Segment e : triangleSides) {
			// true if intersection exists
			bool collision = side.intersection(e, &intersectionPoint);
			if (collision) {
                // check to see if this point is already accounted for by corners
                // or by triangle vertices; if it isn't exactly equal it won't contribute to area
                // (and the lack of exact equality is likely due to floating point error)
                if (!approxEqual(intersectionPoint, corner) && !approxEqual(intersectionPoint, corners[(i+start+1)%4])) {
                    bool isVertex = false;
                    for(Point *tVertex : t.vertices) {
                        if (approxEqual(intersectionPoint, *tVertex)) {
                            isVertex = true;
                        }
                    }
                    if (!isVertex) {
						sideIntersections[intersectNum] = intersectionPoint;
						intersectNum++;
                    }
                }
            }
		}
		/*
		if(intersectNum > 2) {
			printf("INCORRECT INTERSECTION NUM: %d\n", intersectNum);
			for(int k = 0; k < intersectNum; k++) {
				printf("(%f, %f)\n", sideIntersections[k].getX(), sideIntersections[k].getY());
			}
			printf("END INCORRECT\n");
		}
		*/
        // note a triangle can intersect a given side at most twice
        assert(intersectNum <= 2);
		// handle normal case where there is only one intersection with this side
        if (intersectNum == 1) {
			boundary[numPoints] = sideIntersections[0];
			numPoints++;
        } else if (intersectNum == 2) {
            double signedArea = Triangle::getSignedArea(&center, &sideIntersections[0], &sideIntersections[1]);
            // if signedArea == 0, sideIntersections must contain two of the same point
            // which means one vertex of the triangle is on the side; this has
			// already been accounted for and shouldn't happen because of vertex check
			if(signedArea != 0) {
				numPoints += 2;
				int nearestInd = (signedArea < 0) ? 1 : 0; // first point of sideIntersections in ccw order
				boundary[numPoints-2] = sideIntersections[nearestInd];
				boundary[numPoints-1] = sideIntersections[1 - nearestInd];
			}
		}
    }
    // check for null pointer
    if (polygon && size) {
        polygon = boundary;
		*size = numPoints;
    }
    return shoelace(boundary, numPoints);
}

__device__ double Pixel::approxArea(Triangle &t, int n) {
	// width of a square in the lattice grid;
	// this ensures n points per side
	double ds = 1.0/(n-1);
	int numPoints = 0; // number of lattice points inside the triangle
	// weight boundary points by 1/2, as in Pick's
	for(int i = 0; i < n; i++) {
		for(int j = 0; j < n; j++) {
			double xval = x - 0.5 + ds * i;
			double yval = y - 0.5 + ds * j;
			bool contains = true; // whether this point is contained
			bool strictly = true;
			// iterate over vertices
			for(int v = 0; v < 3; v++) {
				int w = (v+1)%3; // avoid doing slow computation twice
				// these accesses seem slow but there doesn't seem to be a better way
				double bx = t.vertices[v]->getX() - xval;
				double by = t.vertices[v]->getY() - yval;
				double cx = t.vertices[w]->getX() - xval;
				double cy = t.vertices[w]->getY() - yval;
				double sign = bx * cy - cx * by;
				// branch divergence here :( nothing seems to speed it up?
				if(sign < 0) {
					contains = false;
					strictly = false;
					break;
				}
				if(sign == 0) strictly = false;
			}
			// count boundary points once and interior points twice
			numPoints += contains + strictly;
		}
	}
	// approximate area
	return numPoints / (2.0 * n * n);
}


int pixelRound(double x, int bound) {
	int floor = (int) x;
	if (abs(x - floor) <= 0.5) {
		return floor;
	} else if (x > 0) {
		return min(floor + 1, bound - 1);
	}
	return 0;
}