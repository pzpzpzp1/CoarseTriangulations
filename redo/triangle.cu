#include "hip/hip_runtime.h"
#include "triangle.cuh"

// custom rounding function to support needed pixel rounding

Triangle::Triangle(Point *a, Point *b, Point *c) {
	vertices[0] = a;
	vertices[1] = b;
	vertices[2] = c;
	if(getSignedArea() < 0) { // reverse direction
		vertices[1] = c;
		vertices[2] = b;
	}
}

double Triangle::getSignedArea() {
	double ax = vertices[0]->getX();
	double ay = vertices[0]->getY();
	double bx = vertices[1]->getX();
	double by = vertices[1]->getY();
	double cx = vertices[2]->getX();
	double cy = vertices[2]->getY();
	// determinant of matrix [bx - ax, cx - ax, by - ay, cy - ay] / 2
	return ((bx - ax) * (cy - ay) - (cx - ax) * (by - ay)) / 2;
}

double Triangle::getArea() {
	double signedArea = getSignedArea();
	if (signedArea < 0) {
		return -signedArea;
	}
	return signedArea;
}

double Triangle::dA(int &p, double vx, double vy) {
	// first extract the other two endpoints; note order matters
	Point* edgePoints[2];
	// retrieve in ccw order
	edgePoints[0] = vertices[(p+1)%3];
	edgePoints[1] = vertices[(p+2)%3];
	// change is -velocity dot edge normal of length |e|/2
	Segment opposite(edgePoints[0], edgePoints[1]);
	// get normal to segment
	double nx, ny;
	opposite.scaledNormal(&nx, &ny);
	// return negative of dot product
	return -(vx * nx + vy * ny);
}

double Triangle::gradX(int &p) {
	return dA(p, 1, 0);
}

double Triangle::gradY(int &p) {
	return dA(p, 0, 1);
}

__device__ bool Triangle::contains(Point &p) {
	// p is inside the triangle iff the orientations of the triangles
	// with vertices (vertices[i], vertices[i+1], p) are all ccw
	for(int i = 0; i < 3; i++) {
		if (Triangle::getSignedArea(vertices[i], vertices[(i+1)%3], &p) < 0) {
			return false;
		}
	}
	return true;
}

int Triangle::midVertex() {
	double distances[3];
	for(int i = 0; i < 3; i++) {
		// get length of opposite side
		distances[i] = vertices[(i+1)%3]->distance(*vertices[(i+2)%3]);
	}
	for(int i = 0; i < 3; i++) {
		if(distances[i] >= min(distances[(i+1)%3], distances[(i+2)%3]) && 
			distances[i] <= max(distances[(i+1)%3], distances[(i+2)%3])) return i;
	}
	throw runtime_error("should not get here");
	return -1; // to make compiler happy
}

double Triangle::maxLength() {
	double distance = 0;
	for(int i = 0; i < 3; i++) {
		distance = max(distance, vertices[(i+1)%3]->distance(*vertices[(i+2)%3]));
	}
	return distance;
}

void Triangle::copyVertices(Point *ptrA, Point *ptrB, Point *ptrC) {
	*ptrA = *vertices[0];
	*ptrB = *vertices[1];
	*ptrC = *vertices[2];
}

double Triangle::getSignedArea(Point *a, Point *b, Point *c) {
	double ax = a->getX();
	double ay = a->getY();
	double bx = b->getX();
	double by = b->getY();
	double cx = c->getX();
	double cy = c->getY();
	return ((bx - ax) * (cy - ay) - (cx - ax) * (by - ay)) / 2;
}

ostream& operator<<(ostream& os, const Triangle &t) {
	os << "Triangle ";
	for(Point *ptr : t.vertices) {
		os << *ptr << " ";
	}
	os << "\n";
	return os;
}
